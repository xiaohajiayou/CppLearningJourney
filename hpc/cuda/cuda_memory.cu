#include "hip/hip_runtime.h"

__global__  void add(float* x, float* y, float* z, int n) {

} 


int main() {
    const int n = 1e9;
    const size = n * sizeof(float) * n;
    float *x, *y, *z;
    x = (float*)malloc(sizeof(float) * n);
    y = (float*)malloc(sizeof(float) * n);
    z = (float*)malloc(sizeof(float) * n);

    for(int i = 0; i < n; ++i) {
        x[i] = 10.0;
        y[i] = 20.0;
    }

    float *d_x, *d_y, *d_z;
    hipMalloc((void**)&d_x, size);
    hipMalloc((void**)&d_y, size);
    hipMalloc((void**)&d_z, size);

    hipMemcpy((void*)d_x, (void*)x, size, hipMemcpyHostToDevice);
    hipMemcpy((void*)d_y, (void*)y, size, hipMemcpyHostToDevice);
    
    return 0;
}